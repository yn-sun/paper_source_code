#include "hip/hip_runtime.h"
#include "program.cuh"
namespace cusr {
    namespace program {

        Program crossover_mutation(Program &parent, Program &donor) {
            Program ret;
            auto donor_index = rand_subtree_index_roulette(donor.prefix, true);
            auto parent_index = rand_subtree_index_roulette(parent.prefix, true);

            int length =
                    parent.length - parent_index.second + parent_index.first + donor_index.second - donor_index.first;
            ret.prefix.resize(length);
            if (parent_index.first > 0) {
                std::copy(parent.prefix.begin(), parent.prefix.begin() + parent_index.first, ret.prefix.begin());
            }
            std::copy(donor.prefix.begin() + donor_index.first, donor.prefix.begin() + donor_index.second,
                      ret.prefix.begin() + parent_index.first);
            int tmp_start_pos = length - ((int) parent.prefix.size() - parent_index.second);
            std::copy(parent.prefix.begin() + parent_index.second, parent.prefix.end(),
                      ret.prefix.begin() + tmp_start_pos);
            ret.length = length;
            ret.depth = get_depth_of_prefix(ret.prefix);
            return ret;
        }


        Program
        point_mutation(Program &program, vector<Function> &function_set, pair<float, float> &range, int variable_num) {
            Program ret;
            ret.prefix.assign(program.prefix.begin(), program.prefix.end());
            int pos = gen_rand_int(0, program.length - 1);
            if (ret.prefix[pos].node_type == BINARY_FUNCTION) {
                auto pre_func = ret.prefix[pos].function;
                rand_function(ret.prefix[pos], function_set);
                while (ret.prefix[pos].function == pre_func || ret.prefix[pos].node_type == UNARY_FUNCTION) {
                    rand_function(ret.prefix[pos], function_set);
                }
            } else if (ret.prefix[pos].node_type == UNARY_FUNCTION) {
                auto pre_func = ret.prefix[pos].function;
                rand_function(ret.prefix[pos], function_set);
                while (ret.prefix[pos].function == pre_func || ret.prefix[pos].node_type == BINARY_FUNCTION) {
                    rand_function(ret.prefix[pos], function_set);
                }
            } else if (ret.prefix[pos].node_type == VARIABLE) {
                int pre_var = ret.prefix[pos].variable;
                rand_terminal(ret.prefix[pos], range, variable_num);
                while (ret.prefix[pos].node_type == VARIABLE && ret.prefix[pos].variable == pre_var) {
                    rand_terminal(ret.prefix[pos], range, variable_num);
                }
            } else {
                float pre_const = ret.prefix[pos].constant;
                rand_terminal(ret.prefix[pos], range, variable_num);
                while (ret.prefix[pos].node_type == CONSTANT && ret.prefix[pos].constant == pre_const) {
                    rand_terminal(ret.prefix[pos], range, variable_num);
                }
            }
            ret.length = program.length;
            ret.depth = program.depth;
            return ret;
        }


        Program hoist_mutation(Program &program) {
            if (program.prefix.size() <= 6) {
                return program;
            }
            auto subtree_index_1 = rand_subtree_index_roulette(program.prefix, false);
            prefix_t tmp(program.prefix.begin() + subtree_index_1.first,
                         program.prefix.begin() + subtree_index_1.second);
            auto subtree_index_2 = rand_subtree_index_roulette(tmp, true);

            while (subtree_index_2.first == 0) {
                subtree_index_2 = rand_subtree_index_roulette(tmp, true);
            }
            Program ret;
            if (subtree_index_1.first > 0) {
                ret.prefix.assign(program.prefix.begin(), program.prefix.begin() + subtree_index_1.first);
            }
            for (int i = subtree_index_2.first; i < subtree_index_2.second; i++) {
                ret.prefix.emplace_back(tmp[i]);
            }

            if (subtree_index_1.second < program.prefix.size()) {
                for (int i = subtree_index_1.second; i < program.prefix.size(); i++) {
                    ret.prefix.emplace_back(program.prefix[i]);
                }
            }

            ret.length = ret.prefix.size();
            ret.depth = get_depth_of_prefix(ret.prefix);
            return ret;
        }


        Program subtree_mutation(Program &program, int depth_of_rand_tree,
                                 pair<float, float> &range, vector<Function> &func_set, int variable_num) {
            prefix_t rand_prefix;
            if (gen_rand_float(0, 1) < 0.5) {
                get_init_prefix(rand_prefix, gen_full_init_tree(depth_of_rand_tree, range, func_set, variable_num));
            } else {
                get_init_prefix(rand_prefix, gen_growth_init_tree(depth_of_rand_tree, range, func_set, variable_num));
            }

            Program temp;
            temp.prefix = rand_prefix;
            return crossover_mutation(program, temp);
        }


        void
        calculate_fitness_cpu(Program *program, const vector<vector<float>> &dataset,
                              const vector<float> &real_value, int data_size,
                              metric_t metric_type) {
            float total_fitness = 0;
            auto *stack = new float[program->depth + 1];
            for (int row = 0; row < data_size; row++) {
                int top = 0;
                for (int i = program->length - 1; i >= 0; i--) {
                    auto &node = program->prefix[i];
                    if (node.node_type == CONSTANT) {
                        stack[top++] = node.constant;
                    } else if (node.node_type == VARIABLE) {
                        stack[top++] = dataset[row][node.variable];
                    } else if (node.node_type == UNARY_FUNCTION) {
                        float var1 = stack[--top];
                        if (node.function == Function::_sin) {
                            stack[top++] = std::sin(var1);
                        } else if (node.function == Function::_cos) {
                            stack[top++] = std::cos(var1);
                        } else if (node.function == Function::_tan) {
                            stack[top++] = std::tan(var1);
                        } else if (node.function == Function::_log) {
                            if (var1 <= 0) {
                                stack[top++] = -1.0f;
                            } else {
                                stack[top++] = std::log(var1);
                            }
                        } else if (node.function == Function::_inv) {
                            if (var1 == 0) {
                                var1 = DELTA;
                            }
                            stack[top++] = 1.0f / var1;
                        }
                    } else {
                        float var1 = stack[--top];
                        float var2 = stack[--top];
                        if (node.function == Function::_add) {
                            stack[top++] = var1 + var2;
                        } else if (node.function == Function::_sub) {
                            stack[top++] = var1 - var2;
                        } else if (node.function == Function::_mul) {
                            stack[top++] = var1 * var2;
                        } else if (node.function == Function::_div) {
                            if (var2 == 0) {
                                var2 = DELTA;
                            }
                            stack[top++] = var1 / var2;
                        } else if (node.function == Function::_max) {
                            stack[top++] = var1 >= var2 ? var1 : var2;
                        } else if (node.function == Function::_min) {
                            stack[top++] = var1 <= var2 ? var1 : var2;
                        }
                    }
                }
                float metric = stack[top - 1] - real_value[row];
                if (metric_type == metric_t::mean_square_error || metric_type == metric_t::root_mean_square_error) {
                    total_fitness += metric * metric;
                } else {
                    total_fitness += metric > 0 ? metric : -metric;
                }
            }
            delete[] stack;
            if (metric_type == root_mean_square_error) {
                program->fitness = std::sqrt(total_fitness / (float) data_size);
            } else {
                program->fitness = total_fitness / (float) data_size;
            }
        }


        int tournament_selection_cpu(vector<Program> &population, int tournament_size, float parsimony_coefficient) {
            int size = population.size();
            int best_index = gen_rand_int(0, size - 1);
            for (int i = 0; i < tournament_size - 1; i++) {
                int rand_index = gen_rand_int(0, size - 1);
                if (population[rand_index].fitness + population[rand_index].length * parsimony_coefficient
                    < population[best_index].fitness + population[best_index].length * parsimony_coefficient) {
                    best_index = rand_index;
                }
            }
            return best_index;
        }


        Program *
        gen_full_init_program(int depth, pair<float, float> &range, vector<Function> &func_set, int variable_num) {
            auto *program = new Program();
            get_init_prefix(program->prefix, gen_full_init_tree(depth, range, func_set, variable_num));
            program->length = program->prefix.size();
            program->depth = get_depth_of_prefix(program->prefix);
            return program;
        }


        Program *
        gen_growth_init_program(int depth, pair<float, float> &range, vector<Function> &func_set, int variable_num) {
            auto *program = new Program();
            while (true) {
                get_init_prefix(program->prefix, gen_growth_init_tree(depth, range, func_set, variable_num));
                program->length = program->prefix.size();
                program->depth = get_depth_of_prefix(program->prefix);
                if (program->length != 1) {
                    break;
                } else {
                    delete program;
                    program = new Program();
                }
            }
            return program;
        }


        Program point_replace_mutation(Program &program, vector<Function> &function_set, pair<float, float> &range,
                                       int variable_num) {
            Program ret;
            ret.prefix.assign(program.prefix.begin(), program.prefix.end());
            for (int pos = 0; pos < program.length; pos++) {
                if (ret.prefix[pos].node_type == BINARY_FUNCTION) {
                    auto pre_func = ret.prefix[pos].function;
                    rand_function(ret.prefix[pos], function_set);
                    while (ret.prefix[pos].function == pre_func || ret.prefix[pos].node_type == UNARY_FUNCTION) {
                        rand_function(ret.prefix[pos], function_set);
                    }
                } else if (ret.prefix[pos].node_type == UNARY_FUNCTION) {
                    auto pre_func = ret.prefix[pos].function;
                    rand_function(ret.prefix[pos], function_set);
                    while (ret.prefix[pos].function == pre_func || ret.prefix[pos].node_type == BINARY_FUNCTION) {
                        rand_function(ret.prefix[pos], function_set);
                    }
                } else if (ret.prefix[pos].node_type == VARIABLE) {
                    int pre_var = ret.prefix[pos].variable;
                    rand_terminal(ret.prefix[pos], range, variable_num);
                    // make sure that the variable is different
                    while (ret.prefix[pos].node_type == VARIABLE && ret.prefix[pos].variable == pre_var) {
                        rand_terminal(ret.prefix[pos], range, variable_num);
                    }
                } else {
                    float pre_const = ret.prefix[pos].constant;
                    rand_terminal(ret.prefix[pos], range, variable_num);
                    // make sure that constant is different
                    while (ret.prefix[pos].node_type == CONSTANT && ret.prefix[pos].constant == pre_const) {
                        rand_terminal(ret.prefix[pos], range, variable_num);
                    }
                }
            }
            ret.length = program.length;
            ret.depth = program.depth;
            return ret;
        }
    }
}