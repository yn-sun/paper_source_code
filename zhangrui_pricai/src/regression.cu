#include "hip/hip_runtime.h"
#include "regression.cuh"

namespace cusr {

    using namespace std;
    using namespace program;
    using namespace fit;

    void RegressionEngine::fit(vector<vector<float>> &dataset, vector<float> &label) {
        this->dataset = dataset;
        this->label = label;
        cusr::program::set_constant_prob(this->p_constant);
        do_fit_init();

        clock_t iter_begin = clock();

        do_population_init();
        update_population_attributes();

        printf("%15s %15s %15s %15s %15s %15s\n",
               "gen", "best fit", "best len", "best dep", "max len", "max dep");
        printf("---------------------------------------------------");
        printf("---------------------------------------------------\n");

        printf("%15d %15.5f %15d %15d %15d %15d\n",
               0, best_program.fitness, best_program.length, best_program.depth, max_length_in_population,
               max_depth_in_population);

        int iter_times = 1;

        while (true) {
            gen_next_generation();
            update_population_attributes();

            printf("%15d %15.5f %15d %15d %15d %15d\n",
                   iter_times, best_program.fitness, best_program.length, best_program.depth, max_length_in_population,
                   max_depth_in_population);

            if (++iter_times >= generations || this->best_program.fitness <= this->stopping_criteria) {
                break;
            }
        }
        this->regress_time_in_sec = (float) (clock() - iter_begin) / (float) CLOCKS_PER_SEC;
        printf("---------------------------------------------------");
        printf("---------------------------------------------------\n");
        cout << "> iteration time: " << regress_time_in_sec << "s" << endl;
        cout << "> best program:   " << prefix_to_infix(best_program.prefix) << endl << endl << endl;

        if (use_gpu) {
            freeDataSetAndLabel(&device_dataset);
        }
    }


    void RegressionEngine::do_fit_init() {
        assert(!dataset.empty() && dataset.size() == label.size());

        this->variable_nums = dataset[0].size();

        if (use_gpu) {
            do_gpu_init();
        }
    }


    void RegressionEngine::do_population_init() {
        this->population.clear();
        if (this->init_method == InitMethod::full) {
            for (int i = 0; i < population_size; i++) {
                int depth = gen_rand_int(init_depth.first, init_depth.second);
                this->population.emplace_back(*gen_full_init_program(depth, const_range, function_set, variable_nums));
            }
        }
        if (this->init_method == InitMethod::growth) {
            for (int i = 0; i < population_size; i++) {
                int depth = gen_rand_int(init_depth.first, init_depth.second);
                this->population.emplace_back(
                        *gen_growth_init_program(depth, const_range, function_set, variable_nums));
            }
        }
        if (this->init_method == InitMethod::half_and_half) {
            // assert(population_size >= 2);
            int full_size = population_size / 2;
            int growth_size = population_size - full_size;
            for (int i = 0; i < full_size; i++) {
                int depth = gen_rand_int(init_depth.first, init_depth.second);
                this->population.emplace_back(*gen_full_init_program(depth, const_range, function_set, variable_nums));
            }
            for (int i = 0; i < growth_size; i++) {
                int depth = gen_rand_int(init_depth.first, init_depth.second);
                this->population.emplace_back(
                        *gen_growth_init_program(depth, const_range, function_set, variable_nums));
            }
        }

        if (use_gpu) {
            calculate_population_fitness_gpu();
        } else {
            calculate_population_fitness_cpu();
        }
    }


    Program RegressionEngine::do_mutation(Program &program) {
        Program ret;
        float rand_float = gen_rand_float(0, 1);
        if (rand_float < p_crossover) {
            int index = tournament_selection_cpu(population, tournament_size, parsimony_coefficient);
            ret = crossover_mutation(program, population[index]);
        } else if (rand_float < p_crossover + p_hoist_mutation) {
            ret = hoist_mutation(program);
        } else if (rand_float < p_crossover + p_hoist_mutation + p_point_mutation) {
            ret = point_mutation(program, function_set, const_range, variable_nums);
        } else if (rand_float < p_crossover + p_hoist_mutation + p_point_mutation + p_subtree_mutation) {
            int rand_int = gen_rand_int(init_depth.first, init_depth.second);
            ret = subtree_mutation(program, rand_int, const_range, function_set, variable_nums);
        } else if (rand_float <
                   p_crossover + p_hoist_mutation + p_point_mutation + p_subtree_mutation + p_point_replace) {
            ret = point_replace_mutation(program, function_set, const_range, variable_nums);
        } else {
            return program;
        }

        ret.depth = get_depth_of_prefix(ret.prefix);

        // hoist until the depth under the specified depth
        while (restrict_depth && ret.depth > max_program_depth) {
            ret = hoist_mutation(ret);
            ret.depth = get_depth_of_prefix(ret.prefix);
        }

        ret.length = ret.prefix.size();
        return ret;
    }


    void RegressionEngine::gen_next_generation() {
        vector<Program> next_gen;

        // elite strategy
        int best_fitness_index = 0;
        for (int i = 1; i < population_size; i++) {
            if (population[i].fitness < population[best_fitness_index].fitness) {
                best_fitness_index = i;
            }
        }
        next_gen.emplace_back(population[best_fitness_index]);

        // selection
        for (int i = 1; i < population_size; i++) {
            int index = tournament_selection_cpu(population, tournament_size, parsimony_coefficient);
            next_gen.emplace_back(do_mutation(population[index]));
        }

        population.assign(next_gen.begin(), next_gen.end());

        // fitness evaluation
        if (use_gpu) {
            calculate_population_fitness_gpu();
        } else {
            calculate_population_fitness_cpu();
        }

    }


    void RegressionEngine::update_population_attributes() {
        int best_fitness_index = 0;
        int max_prefix_length = 0;
        int max_prefix_depth = 0;

        for (int i = 1; i < population_size; i++) {
            if (population[i].fitness < population[best_fitness_index].fitness) {
                best_fitness_index = i;
            }
            if (population[i].length > max_prefix_length) {
                max_prefix_length = population[i].length;
            }
            if (population[i].depth > max_prefix_depth) {
                max_prefix_depth = population[i].depth;
            }
        }

        this->best_program = population[best_fitness_index];
        this->max_length_in_population = max_prefix_length;
        this->max_depth_in_population = max_prefix_depth;
        this->best_program_in_each_gen.emplace_back(this->best_program);
    }


    void RegressionEngine::calculate_population_fitness_cpu() {
        for (int i = 0; i < population_size; i++) {
            calculate_fitness_cpu(&population[i], dataset, label, dataset.size(), this->metric);
        }
    }


    void RegressionEngine::calculate_population_fitness_gpu() {
        int blockNum = (dataset.size() - 1) / THREAD_PER_BLOCK + 1;
        calculatePopulationFitness(this->device_dataset, blockNum, population, this->metric);
    }


    void RegressionEngine::do_gpu_init() {
        copyDatasetAndLabel(&device_dataset, dataset, label);
    }


    RegressionEngine::~RegressionEngine() {
        freeDataSetAndLabel(&this->device_dataset);
    }
}