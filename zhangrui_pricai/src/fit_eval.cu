#include "hip/hip_runtime.h"
#include "fit_eval.cuh"

namespace cusr {
    namespace fit {

        using namespace program;
        using namespace std;

        void copyDatasetAndLabel(GPUDataset *dataset_struct, vector<vector<float>> &dataset, vector<float> &label) {
            dataset_struct->dataset_size = dataset.size();

            // format dataset into column-major
            int data_size = dataset.size();
            int variable_num = dataset[0].size();

            vector<float> device_dataset;

            for (int i = 0; i < variable_num; i++) {
                for (int j = 0; j < data_size; j++) {
                    device_dataset.emplace_back(dataset[j][i]);
                }
            }

            // copy dataset
            float *device_dataset_arr;
            size_t dataset_pitch;
            hipMallocPitch((void **) &device_dataset_arr, &dataset_pitch, sizeof(float) * data_size, variable_num);
            hipMemcpy2D(device_dataset_arr, dataset_pitch, thrust::raw_pointer_cast(device_dataset.data()),
                         sizeof(float) * data_size, sizeof(float) * data_size, variable_num, hipMemcpyHostToDevice);

            dataset_struct->dataset_pitch = dataset_pitch;
            dataset_struct->dataset = device_dataset_arr;

            // copy label set
            float *device_label_arr;
            hipMalloc((void **) &device_label_arr, sizeof(float) * data_size);
            hipMemcpy(device_label_arr, thrust::raw_pointer_cast(label.data()), sizeof(float) * data_size,
                       hipMemcpyHostToDevice);

            dataset_struct->label = device_label_arr;
        }


        void freeDataSetAndLabel(GPUDataset *dataset_struct) {
            hipFree(dataset_struct->dataset);
            hipFree(dataset_struct->label);
        }


        __constant__ float d_nodeValue[MAX_PREFIX_LEN];
        __constant__ float d_nodeType[MAX_PREFIX_LEN];


#define S_OFF THREAD_PER_BLOCK * (DEPTH + 1) * blockIdx.x + top * THREAD_PER_BLOCK + threadIdx.x


        __global__ void
        calFitnessGPU_MSE(int len, float *ds, int dsPitch, float *label, float *stack, float *result, int dataset_size) {
            extern __shared__ float shared[];
            shared[threadIdx.x] = 0;
            // each thread is responsible for one datapoint
            int dataset_no = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;
            if (dataset_no < dataset_size) {
                int top = 0;
                for (int i = len - 1; i >= 0; i--) {
                    int node_type = d_nodeType[i];
                    float node_value = d_nodeValue[i];

                    if (node_type == CONSTANT) {
                        stack[S_OFF] = node_value;
                        top++;
                    } else if (node_type == VARIABLE) {
                        int var_num = node_value;
                        stack[S_OFF] = ((float *) ((char *) ds + var_num * dsPitch))[dataset_no];
                        top++;
                    } else if (node_type == UNARY_FUNCTION) {
                        int function = node_value;
                        top--;
                        float var1 = stack[S_OFF];
                        if (function == SIN_SIGN) {
                            stack[S_OFF] = std::sin(var1);
                            top++;
                        } else if (function == COS_SIGN) {
                            stack[S_OFF] = std::cos(var1);
                            top++;
                        } else if (function == TAN_SIGN) {
                            stack[S_OFF] = std::tan(var1);
                            top++;
                        } else if (function == LOG_SIGN) {
                            if (var1 <= 0) {
                                stack[S_OFF] = -1.0f;
                                top++;
                            } else {
                                stack[S_OFF] = std::log(var1);
                                top++;
                            }
                        } else if (function == INV_SIGN) {
                            if (var1 == 0) {
                                var1 = DELTA;
                            }
                            stack[S_OFF] = 1.0f / var1;
                            top++;
                        }
                    } else // if (node_type == BINARY_FUNCTION)
                    {
                        int function = node_value;
                        top--;
                        float var1 = stack[S_OFF];
                        top--;
                        float var2 = stack[S_OFF];
                        if (function == ADD_SIGN) {
                            stack[S_OFF] = var1 + var2;
                            top++;
                        } else if (function == SUB_SIGN) {
                            stack[S_OFF] = var1 - var2;
                            top++;
                        } else if (function == MUL_SIGN) {
                            stack[S_OFF] = var1 * var2;
                            top++;
                        } else if (function == DIV_SIGN) {
                            if (var2 == 0) {
                                var2 = DELTA;
                            }
                            stack[S_OFF] = var1 / var2;
                            top++;
                        } else if (function == MAX_SIGN) {
                            stack[S_OFF] = var1 >= var2 ? var1 : var2;
                            top++;
                        } else if (function == MIN_SIGN) {
                            stack[S_OFF] = var1 <= var2 ? var1 : var2;
                            top++;
                        }
                    }
                }
                top--;
                float prefix_value = stack[S_OFF];
                float label_value = label[dataset_no];
                float loss = prefix_value - label_value;
                float fitness = loss * loss;
                shared[threadIdx.x] = fitness;
            }
            __syncthreads();

#if THREAD_PER_BLOCK >= 1024
            if (threadIdx.x < 512) { shared[threadIdx.x] += shared[threadIdx.x + 512]; }
            __syncthreads();
#endif
#if THREAD_PER_BLOCK >= 512
            if (threadIdx.x < 256) { shared[threadIdx.x] += shared[threadIdx.x + 256]; }
            __syncthreads();
#endif
            if (threadIdx.x < 128) { shared[threadIdx.x] += shared[threadIdx.x + 128]; }
            __syncthreads();
            if (threadIdx.x < 64) { shared[threadIdx.x] += shared[threadIdx.x + 64]; }
            __syncthreads();
            if (threadIdx.x < 32) { shared[threadIdx.x] += shared[threadIdx.x + 32]; }
            if (threadIdx.x < 16) { shared[threadIdx.x] += shared[threadIdx.x + 16]; }
            if (threadIdx.x < 8) { shared[threadIdx.x] += shared[threadIdx.x + 8]; }
            if (threadIdx.x < 4) { shared[threadIdx.x] += shared[threadIdx.x + 4]; }
            if (threadIdx.x < 2) { shared[threadIdx.x] += shared[threadIdx.x + 2]; }
            if (threadIdx.x < 1) {
                shared[threadIdx.x] += shared[threadIdx.x + 1];
//                result[blockIdx.x] = shared[0] / THREAD_PER_BLOCK;
                result[blockIdx.x] = shared[0];
            }
        }


        __global__ void
        calFitnessGPU_MAE(int len, float *ds, int dsPitch, float *label, float *stack, float *result, int dataset_size) {
            extern __shared__ float shared[];
            shared[threadIdx.x] = 0;
            int dataset_no = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;
            if (dataset_no < dataset_size) {
                int top = 0;
                for (int i = len - 1; i >= 0; i--) {
                    int node_type = d_nodeType[i];
                    float node_value = d_nodeValue[i];

                    if (node_type == CONSTANT) {
                        stack[S_OFF] = node_value;
                        top++;
                    } else if (node_type == VARIABLE) {
                        int var_num = node_value;
                        stack[S_OFF] = ((float *) ((char *) ds + var_num * dsPitch))[dataset_no];
                        top++;
                    } else if (node_type == UNARY_FUNCTION) {
                        int function = node_value;
                        top--;
                        float var1 = stack[S_OFF];
                        if (function == SIN_SIGN) {
                            stack[S_OFF] = std::sin(var1);
                            top++;
                        } else if (function == COS_SIGN) {
                            stack[S_OFF] = std::cos(var1);
                            top++;
                        } else if (function == TAN_SIGN) {
                            stack[S_OFF] = std::tan(var1);
                            top++;
                        } else if (function == LOG_SIGN) {
                            if (var1 <= 0) {
                                stack[S_OFF] = -1.0f;
                                top++;
                            } else {
                                stack[S_OFF] = std::log(var1);
                                top++;
                            }
                        } else if (function == INV_SIGN) {
                            if (var1 == 0) {
                                var1 = DELTA;
                            }
                            stack[S_OFF] = 1.0f / var1;
                            top++;
                        }
                    } else {
                        int function = node_value;
                        top--;
                        float var1 = stack[S_OFF];
                        top--;
                        float var2 = stack[S_OFF];
                        if (function == ADD_SIGN) {
                            stack[S_OFF] = var1 + var2;
                            top++;
                        } else if (function == SUB_SIGN) {
                            stack[S_OFF] = var1 - var2;
                            top++;
                        } else if (function == MUL_SIGN) {
                            stack[S_OFF] = var1 * var2;
                            top++;
                        } else if (function == DIV_SIGN) {
                            if (var2 == 0) {
                                var2 = DELTA;
                            }
                            stack[S_OFF] = var1 / var2;
                            top++;
                        } else if (function == MAX_SIGN) {
                            stack[S_OFF] = var1 >= var2 ? var1 : var2;
                            top++;
                        } else if (function == MIN_SIGN) {
                            stack[S_OFF] = var1 <= var2 ? var1 : var2;
                            top++;
                        }
                    }
                }
                top--;
                float prefix_value = stack[S_OFF];
                float label_value = label[dataset_no];
                float loss = prefix_value - label_value;
                float fitness = loss >= 0 ? loss : -loss;
                shared[threadIdx.x] = fitness;
            }
            __syncthreads();

#if THREAD_PER_BLOCK >= 1024
            if (threadIdx.x < 512) { shared[threadIdx.x] += shared[threadIdx.x + 512]; }
            __syncthreads();
#endif

#if THREAD_PER_BLOCK >= 512
            if (threadIdx.x < 256) { shared[threadIdx.x] += shared[threadIdx.x + 256]; }
            __syncthreads();
#endif

            if (threadIdx.x < 128) { shared[threadIdx.x] += shared[threadIdx.x + 128]; }
            __syncthreads();
            if (threadIdx.x < 64) { shared[threadIdx.x] += shared[threadIdx.x + 64]; }
            __syncthreads();
            if (threadIdx.x < 32) { shared[threadIdx.x] += shared[threadIdx.x + 32]; }
            if (threadIdx.x < 16) { shared[threadIdx.x] += shared[threadIdx.x + 16]; }
            if (threadIdx.x < 8) { shared[threadIdx.x] += shared[threadIdx.x + 8]; }
            if (threadIdx.x < 4) { shared[threadIdx.x] += shared[threadIdx.x + 4]; }
            if (threadIdx.x < 2) { shared[threadIdx.x] += shared[threadIdx.x + 2]; }
            if (threadIdx.x < 1) {
                shared[threadIdx.x] += shared[threadIdx.x + 1];
                result[blockIdx.x] = shared[0];
            }
        }


        float *mallocStack(int blockNum) {
            float *stack;
            // allocate stack space, the size of which = sizeof(float) * THREAD_PER_BLOCK * (maxDepth + 1)
            hipMalloc((void **) &stack, sizeof(float) * THREAD_PER_BLOCK * (DEPTH + 1) * blockNum);
            return stack;
        }


        void calSingleProgram(GPUDataset &dataset, int blockNum, Program &program,
                              float *stack, float *result, float *h_res, metric_t metric) {
            // --------- restrict the length of prefix ---------
            assert(program.length < MAX_PREFIX_LEN);
            // -------------------------------------------------

            // -------- copy to constant memory --------
            float h_nodeValue[MAX_PREFIX_LEN];
            float h_nodeType[MAX_PREFIX_LEN];
            for (int i = 0; i < program.length; i++) {
                char type = program.prefix[i].node_type;
                h_nodeType[i] = type;
                if (type == 'c') {
                    h_nodeValue[i] = program.prefix[i].constant;
                } else if (type == 'v') {
                    h_nodeValue[i] = program.prefix[i].variable;
                } else // if (type == 'u' || type == 'v')
                {
                    h_nodeValue[i] = program.prefix[i].function;
                }
            }
            hipMemcpyToSymbol(HIP_SYMBOL(d_nodeValue), h_nodeValue, sizeof(float) * program.length);
            hipMemcpyToSymbol(HIP_SYMBOL(d_nodeType), h_nodeType, sizeof(float) * program.length);

            // -------- calculation and synchronization --------
            if (metric == metric_t::mean_absolute_error) {
                calFitnessGPU_MAE<<<blockNum, THREAD_PER_BLOCK, sizeof(float) * THREAD_PER_BLOCK>>>
                        (program.length, dataset.dataset, dataset.dataset_pitch, dataset.label, stack, result, dataset.dataset_size);
                hipDeviceSynchronize();
            } else if (metric == metric_t::mean_square_error || metric == metric_t::root_mean_square_error) {
                calFitnessGPU_MSE<<<blockNum, THREAD_PER_BLOCK, sizeof(float) * THREAD_PER_BLOCK >>>
                        (program.length, dataset.dataset, dataset.dataset_pitch, dataset.label, stack, result, dataset.dataset_size);
                hipDeviceSynchronize();
            }

            // -------- reduction on the result --------
            hipMemcpy(h_res, result, sizeof(float) * blockNum, hipMemcpyDeviceToHost);
            float ans = 0;
            for (int i = 0; i < blockNum; i++) {
                ans += h_res[i];
            }
            if (metric == metric_t::mean_absolute_error || metric == metric_t::mean_square_error) {
                program.fitness = ans / (float) dataset.dataset_size;
            } else if (metric == metric_t::root_mean_square_error) {
                program.fitness = std::sqrt(ans / (float) dataset.dataset_size);
            }
        }


        void
        calculatePopulationFitness(GPUDataset &dataset, int blockNum, vector<Program> &population, metric_t metric) {
            // allocate space for result
            float *result;
            hipMalloc((void **) &result, sizeof(float) * blockNum);

            // allocate stack space
            float *stack = mallocStack(blockNum);

            // save result and do CPU side reduction
            float *h_res = new float[blockNum];

            // evaluate fitness for each program in the population
            for (int i = 0; i < population.size(); i++) {
                calSingleProgram(dataset, blockNum, population[i], stack, result, h_res, metric);
            }

            // free memory space
            hipFree(result);
            hipFree(stack);
            delete[] h_res;
        }
    }
}