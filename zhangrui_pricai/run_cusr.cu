
#include <hip/hip_runtime.h>
//#include "include/cusr.h"
//
//using namespace std;
//using namespace cusr;
//
//// we consider approximating Pagie polynomial,
//// which is defined by: f(x, y) = 1 / (1 + x ^ (-4)) + 1 / (1 + y ^ (-4))
//vector<vector<float>> dataset;
//vector<float> real_value;
//
//
//void gen_dataset() {
//    cout << "gen dataset" << endl;
//    for (int i = 0; i < 2048 * 2048; i++) {
//        float x0 = cusr::program::gen_rand_float(-5, 5);
//        float x1 = cusr::program::gen_rand_float(-5, 5);
//        dataset.push_back({x0, x1});
//        real_value.push_back(x0 * x0 * x0 * x0 / (x0 * x0 * x0 * x0 + 1) + x1 * x1 * x1 * x1 / (x1 * x1 * x1 * x1 + 1));
//    }
//    cout << "gen dataset finish" << endl;
//}
//
//
//int main() {
//    gen_dataset();
//    cusr::RegressionEngine reg;
//    reg.function_set = { _add, _cos, _sub, _div, _tan, _mul, _sin };
//    reg.use_gpu = true;            // performing GPU acceleration -- much faster than CPU
//    reg.max_program_depth = 10;    // better less than 20 --
//                                   // or may cause overflow due to the limitation of the length of prefix (less than 2048)
//    reg.population_size = 50;
//    reg.generations = 50;
//    reg.parsimony_coefficient = 0;            // this param prevents program from bloating  -- derived from "gplearn"
//    reg.const_range = {-5, 5};      // the range of the constant of each program
//    reg.init_depth = {4, 10};       // the range of init depth of the expression tree
//    reg.init_method = init_t::half_and_half;   // ramped half-and-half is recommended
//    reg.tournament_size = 3;                   // it only supports tournament selection
//    reg.metric = metric_t::root_mean_square_error; // also support MAE error and MSE error
//    reg.fit(dataset, real_value); // do training
//
//    // after training
//    cout << "execution time: " << reg.regress_time_in_sec << endl;
//    cout << "best fitness  : " << reg.best_program.fitness << endl;
//    // optimal program
//    cout << "best program (in prefix):  " << cusr::program::prefix_to_string(reg.best_program.prefix) << endl;
//    cout << "best program (in infix) :  " << cusr::program::prefix_to_infix(reg.best_program.prefix) << endl;
//    return 0;
//}